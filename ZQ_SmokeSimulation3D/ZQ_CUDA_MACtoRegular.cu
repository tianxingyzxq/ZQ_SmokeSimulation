#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_MAC_TO_REGULAR_CU_
#define _ZQ_CUDA_MAC_TO_REGULAR_CU_

#include "ZQlibCudaDefines.cuh"
#include "ZQ_CUDA_MACtoRegular.cuh"

namespace ZQ_CUDA_MACtoRegular
{
	__global__
	void MAC_to_Regular4_Kernel(const float* mac_u, const float* mac_v, const float* mac_w, float4* vel4, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y >= height)
			return;

		for (int z = 0; z < depth; z++)
		{
			int offset = z*height*width + y*width + x;
			float u = 0.5f*(mac_u[z*height*(width + 1) + y*(width + 1) + x] + mac_u[z*height*(width + 1) + y*(width + 1) + x + 1]);
			float v = 0.5f*(mac_v[z*(height + 1)*width + y*width + x] + mac_v[z*(height + 1)*width + (y + 1)*width + x]);
			float w = 0.5f*(mac_w[z*height*width + y*width + x] + mac_w[(z + 1)*height*width + y*width + x]);
			vel4[offset] = make_float4(u, v, w, 0);
		}
	}

	__global__
		void Regular4_to_MAC_u_Kernel(const float4* vel4, float* mac_u, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x > width || y >= height)
			return;

		for (int z = 0; z < depth; z++)
		{
			if (x == 0)
				mac_u[z*height*(width + 1) + y*(width + 1) + x] = vel4[z*height*width + y*width + x].x;
			else if (x == width)
				mac_u[z*height*(width + 1) + y*(width + 1) + x] = vel4[z*height*width + y*width + x - 1].x;
			else
				mac_u[z*height*(width + 1) + y*(width + 1) + x] = 0.5*(vel4[z*height*width + y*width + x].x + vel4[z*height*width + y*width + x - 1].x);
		}
	}

	__global__
		void Regular4_to_MAC_v_Kernel(const float4* vel4, float* mac_v, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y > height)
			return;

		for (int z = 0; z < depth; z++)
		{
			if (y == 0)
				mac_v[z*(height + 1)*width + y*width + x] = vel4[z*height*width + y*width + x].y;
			else if (y == height)
				mac_v[z*(height + 1)*width + y*width + x] = vel4[z*height*width + (y - 1)*width + x].y;
			else
				mac_v[z*(height + 1)*width + y*width + x] = 0.5*(vel4[z*height*width + y*width + x].y + vel4[z*height*width + (y - 1)*width + x].y);
		}
	}

	__global__
		void Regular4_to_MAC_w_Kernel(const float4* vel4, float* mac_w, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y >= height)
			return;

		mac_w[y*width + x] = vel4[y*width + x].z;
		mac_w[depth*height*width + y*width + x] = vel4[(depth - 1)*height*width + y*width + x].z;
		for (int z = 1; z < depth; z++)
		{
			mac_w[z*height*width + y*width + x] = 0.5*(vel4[z*height*width + y*width + x].z + vel4[(z - 1)*height*width + y*width + x].z);
		}
	}

	/******************/
	
	void cu_MAC_to_Regular4(const float* mac_u, const float* mac_v, const float* mac_w, float4* vel4, int width, int height, int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);

		MAC_to_Regular4_Kernel<<<gridSize,blockSize>>>(mac_u,mac_v,mac_w,vel4,width,height,depth);
	}

	void cu_Regular4_to_MAC(const float4* vel4, float* mac_u, float* mac_v, float* mac_w, int width, int height, int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		Regular4_to_MAC_u_Kernel<<<u_gridSize,blockSize>>>(vel4,mac_u,width,height,depth);
		Regular4_to_MAC_v_Kernel<<<v_gridSize,blockSize>>>(vel4,mac_v,width,height,depth);
		Regular4_to_MAC_w_Kernel<<<w_gridSize,blockSize>>>(vel4,mac_w,width,height,depth);
	}
}

#endif