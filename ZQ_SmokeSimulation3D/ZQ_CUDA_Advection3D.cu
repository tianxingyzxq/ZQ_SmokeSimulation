#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_ADVECTION_3D_CU_
#define _ZQ_CUDA_ADVECTION_3D_CU_

#include "ZQ_CUDA_Advection3D.h"
#include "ZQ_CUDA_Advection3D.cuh"
#include "ZQ_CUDA_MACtoRegular.cuh"

namespace ZQ_CUDA_Advection3D
{
	texture<float4,3,hipReadModeElementType> tex_velocity_regular;
	texture<float,3,hipReadModeElementType> tex_velocity_MAC_u;
	texture<float,3,hipReadModeElementType> tex_velocity_MAC_v;
	texture<float,3,hipReadModeElementType> tex_velocity_MAC_w;
	texture<unsigned char,3,hipReadModeElementType> tex_occupy;
	texture<float,3,hipReadModeElementType> tex_inputVelocity_MAC_u;
	texture<float,3,hipReadModeElementType> tex_inputVelocity_MAC_v;
	texture<float,3,hipReadModeElementType> tex_inputVelocity_MAC_w;
	texture<float4,3,hipReadModeElementType> tex_inputVelocity_regular;
	texture<float,3,hipReadModeElementType> tex_temperature; 
	texture<float,3,hipReadModeElementType> tex_density; 

	unsigned h_width;
	unsigned h_height;
	unsigned h_depth;
	unsigned int h_steps;
	float h_voxelSize;
	float h_deltatt;
	
	__constant__ unsigned int d_width;
	__constant__ unsigned int d_height;
	__constant__ unsigned int d_depth;
	__constant__ unsigned int d_steps;
	__constant__ float d_voxelSize;
	__constant__ float d_deltatt;
	
	/****************************************************************************************/
	
	__global__ 
	void Velocity_Negative_Kernel(float* in_out_data, int len)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		if(x >= len)
			return;
		in_out_data[x] = -in_out_data[x];
	}

	__global__ 
	void Velocity_Negative_4channels_Kernel(float4* in_out_data, int len)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		if(x >= len)
			return;
		in_out_data[x] = -in_out_data[x];
	}

	__global__ 
	void Input_Increment_Kernel(float* input, const float* input_star, int len)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		if(x >= len)
			return;

		input[x] = input[x]*1.5 - input_star[x]*0.5;
	}

	__global__ 
	void Input_Increment_4channels_Kernel(float4* input, const float4* input_star, int len)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		if(x >= len)
			return;

		input[x] = input[x]*1.5 - input_star[x]*0.5;
	}

	__global__ 
	void Advect_Velocity_inRegular_outRegular_Kernel(float4 * d_output)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y >= d_height)
			return;
			
		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,y+0.5f,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
			float4 lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = velCoord;
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height&& pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);

				lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord = make_float3(lastpos.x/d_width,lastpos.y/d_height,lastpos.z/d_depth);
			float4 tempvel = tex3D(tex_inputVelocity_regular,out_coord.x,out_coord.y,out_coord.z);
			d_output[z*d_height*d_width+y*d_width+x] = tempvel;
		}
	}

	__global__ 
	void Advect_Velocity_inRegular_outMAC_u_Kernel(float * d_mac_u)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x > d_width || y >= d_height)
			return;

		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4((float)x,y+0.5f,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
			float4 lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = velCoord;
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);

				lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord = make_float3(lastpos.x/d_width,lastpos.y/d_height,lastpos.z/d_depth);
			float4 tempvel = tex3D(tex_inputVelocity_regular,out_coord.x,out_coord.y,out_coord.z);
			d_mac_u[z*d_height*(d_width+1)+y*(d_width+1)+x] = tempvel.x;
		}
	}

	__global__
	void Advect_Velocity_inRegular_outMAC_v_Kernel(float * d_mac_v)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y > d_height)
			return;

		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,(float)y,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
			float4 lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = velCoord;
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);

				lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord = make_float3(lastpos.x/d_width,lastpos.y/d_height,lastpos.z/d_depth);
			float4 tempvel = tex3D(tex_inputVelocity_regular,out_coord.x,out_coord.y,out_coord.z);
			d_mac_v[z*(d_height+1)*d_width+y*d_width+x] = tempvel.y;
		}
	}
	
	__global__
	void Advect_Velocity_inRegular_outMAC_w_Kernel(float * d_mac_w)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y >= d_height)
			return;

		for(int z = 0;z <= d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,y+0.5f,(float)z,0.0f);
			float4 lastpos = pos;
			float3 velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
			float4 lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = velCoord;
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);

				lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord = make_float3(lastpos.x/d_width,lastpos.y/d_height,lastpos.z/d_depth);
			float4 tempvel = tex3D(tex_inputVelocity_regular,out_coord.x,out_coord.y,out_coord.z);
			d_mac_w[z*d_height*d_width+y*d_width+x] = tempvel.z;
		}
	}

	__global__
	void Advect_Velocity_inMAC_outMAC_u_Kernel(float * d_mac_u)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x > d_width || y >= d_height)
			return;

		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4((float)x,y+0.5f,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
			float3 velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
			float3 velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));
			float4 lastvel = make_float4(
				tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
				tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
				tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
				velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
				velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));

				lastvel = make_float4(
					tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
					tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
					tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord_u = make_float3((lastpos.x+0.5f)/(d_width+1),lastpos.y/d_height,lastpos.z/d_depth);
			float tempvel = tex3D(tex_inputVelocity_MAC_u,out_coord_u.x,out_coord_u.y,out_coord_u.z);
			d_mac_u[z*d_height*(d_width+1)+y*(d_width+1)+x] = tempvel;
		}
	}

	__global__
	void Advect_Velocity_inMAC_outMAC_v_Kernel(float * d_mac_v)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y > d_height)
			return;

		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,(float)y,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
			float3 velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
			float3 velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));
			float4 lastvel = make_float4(
				tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
				tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
				tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
				velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
				velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));

				lastvel = make_float4(
					tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
					tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
					tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord_v = make_float3(lastpos.x/d_width,(lastpos.y+0.5f)/(d_height+1),lastpos.z/d_depth);
			float tempvel = tex3D(tex_inputVelocity_MAC_v,out_coord_v.x,out_coord_v.y,out_coord_v.z);
			d_mac_v[z*(d_height+1)*d_width+y*d_width+x] = tempvel;
		}
	}
	
	__global__
	void Advect_Velocity_inMAC_outMAC_w_Kernel(float * d_mac_w)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y >= d_height)
			return;

		for(int z = 0;z <= d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,y+0.5f,(float)z,0.0f);
			float4 lastpos = pos;
			float3 velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
			float3 velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
			float3 velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));
			float4 lastvel = make_float4(
				tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
				tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
				tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
				velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
				velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));

				lastvel = make_float4(
					tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
					tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
					tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord_w = make_float3(lastpos.x/d_width,lastpos.y/d_height,(lastpos.z+0.5f)/(d_depth+1));
			float tempvel = tex3D(tex_inputVelocity_MAC_w,out_coord_w.x,out_coord_w.y,out_coord_w.z);
			d_mac_w[z*d_height*d_width+y*d_width+x] = tempvel;
		}
	}
		
	__global__ 
	void Advect_Scalar_Regular_Velocity_Kernel(float* d_output_temperature, float* d_output_density)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y >= d_height)
			return;

		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,y+0.5f,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
			float4 lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = velCoord;
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);

				lastvel = tex3D(tex_velocity_regular,velCoord.x,velCoord.y,velCoord.z);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord = make_float3(lastpos.x/d_width,lastpos.y/d_height,lastpos.z/d_depth);
			float temperature = tex3D(tex_temperature,out_coord.x,out_coord.y,out_coord.z);
			float density = tex3D(tex_density,out_coord.x,out_coord.y,out_coord.z);
			d_output_temperature[z*d_height*d_width+y*d_width+x] = temperature;
			d_output_density[z*d_height*d_width+y*d_width+x] = density;
		}
	}

	__global__ 
	void Advect_Scalar_MAC_Velocity_Kernel(float* d_output_temperature, float* d_output_density)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if(x >= d_width || y >= d_height)
			return;

		for(int z = 0;z < d_depth;z++)
		{
			float4 pos = make_float4(x+0.5f,y+0.5f,z+0.5f,0.0f);
			float4 lastpos = pos;
			float3 velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
			float3 velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
			float3 velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));
			float4 lastvel = make_float4(
				tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
				tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
				tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);

			unsigned int istep = 0;
			do 
			{
				float3 occupyCoord = make_float3(pos.x/d_width,pos.y/d_height,pos.z/d_depth);
				if(!(pos.x >= 0 && pos.x <= d_width && pos.y >= 0 && pos.y <= d_height && pos.z >= 0 && pos.z <= d_depth))
					break;
				if(tex3D(tex_occupy,occupyCoord.x,occupyCoord.y,occupyCoord.z) != 0)
					break;

				lastpos = pos;
				pos -= lastvel * d_deltatt / d_voxelSize;
				velCoord_u = make_float3((pos.x+0.5f)/(d_width+1),pos.y/d_height,pos.z/d_depth);
				velCoord_v = make_float3(pos.x/d_width,(pos.y+0.5f)/(d_height+1),pos.z/d_depth);
				velCoord_w = make_float3(pos.x/d_width,pos.y/d_height,(pos.z+0.5f)/(d_depth+1));

				lastvel = make_float4(
					tex3D(tex_velocity_MAC_u,velCoord_u.x,velCoord_u.y,velCoord_u.z),
					tex3D(tex_velocity_MAC_v,velCoord_v.x,velCoord_v.y,velCoord_v.z),
					tex3D(tex_velocity_MAC_w,velCoord_w.x,velCoord_w.y,velCoord_w.z),0.0f);
				istep ++;
			} while (istep < d_steps);

			float3 out_coord = make_float3(lastpos.x/d_width,lastpos.y/d_height,lastpos.z/d_depth);
			float temperature = tex3D(tex_temperature,out_coord.x,out_coord.y,out_coord.z);
			float density = tex3D(tex_density,out_coord.x,out_coord.y,out_coord.z);
			d_output_temperature[z*d_height*d_width+y*d_width+x] = temperature;
			d_output_density[z*d_height*d_width+y*d_width+x] = density;
		}
	}

	__global__ 
	void Apply_Advect_Velocity_Result_Open_u_Kernel(const float* adv_u, const bool* occupy, float* u, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x > width || y >= height)
			return;

		int offset = y*width+x;
		int u_offset = y*(width+1)+x;
		for (int z = 0; z < depth; z++)
		{
			if(x == 0)
			{
				if(!occupy[offset])
					u[u_offset] = adv_u[u_offset];
			}
			else if(x == width)
			{
				if(!occupy[offset-1])
					u[u_offset] = adv_u[u_offset];
			}
			else
			{
				if(!occupy[offset] && !occupy[offset-1])
					u[u_offset] = adv_u[u_offset];
			}
			offset += height*width;
			u_offset += height*(width+1);
		}
	}

	__global__ 
	void Apply_Advect_Velocity_Result_Open_v_Kernel(const float* adv_v, const bool* occupy, float* v, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y > height)
			return;

		int offset = y*width+x;
		int v_offset = y*width+x;
		for (int z = 0; z < depth; z++)
		{
			if(y == 0)
			{
				if(!occupy[offset])
					v[v_offset] = adv_v[v_offset];
			}
			else if(y == height)
			{
				if(!occupy[offset-width])
					v[v_offset] = adv_v[v_offset];
			}
			else
			{
				if(!occupy[offset] && !occupy[offset-width])
					v[v_offset] = adv_v[v_offset];
			}
			offset += height*width;
			v_offset += (height+1)*width;
		}
	}

	__global__ 
	void Apply_Advect_Velocity_Result_Open_w_Kernel(const float* adv_w, const bool* occupy, float* w, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y >= height)
			return;

		int offset = y*width+x;
		int w_offset = y*width+x;
		if(!occupy[offset])
			w[w_offset] = adv_w[w_offset];
		offset += height*width;
		w_offset += height*width;

		for (int z = 1; z < depth; z++)
		{
			if(!occupy[offset] && !occupy[offset-height*width])
				w[w_offset] = adv_w[w_offset];
			offset += height*width;
			w_offset += height*width;
		}

		if(!occupy[offset-height*width])
			w[w_offset] = adv_w[w_offset];
	}

	__global__ 
	void Apply_Advect_Velocity_Result_Closed_u_Kernel(const float* adv_u, const bool* occupy, float* u, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x == 0 || x >= width || y >= height)
			return;

		int offset = y*width+x;
		int u_offset = y*(width+1)+x;
		for (int z = 0; z < depth; z++)
		{
			if(!occupy[offset] && !occupy[offset-1])
				u[u_offset] = adv_u[u_offset];
			offset += height*width;
			u_offset += height*(width+1);
		}
	}

	__global__ 
	void Apply_Advect_Velocity_Result_Closed_v_Kernel(const float* adv_v, const bool* occupy, float* v, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y == 0 || y >= height)
			return;

		int offset = y*width+x;
		int v_offset = y*width+x;
		for (int z = 0; z < depth; z++)
		{
			if(!occupy[offset] && !occupy[offset-width])
				v[v_offset] = adv_v[v_offset];
			offset += height*width;
			v_offset += (height+1)*width;
		}
	}

	__global__ 
	void Apply_Advect_Velocity_Result_Closed_w_Kernel(const float* adv_w, const bool* occupy, float* w, int width, int height, int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x + tx;
		int y = by*blockDim.y + ty;
		if (x >= width || y >= height)
			return;

		int offset = y*width+x;
		int w_offset = y*width+x;
		offset += height*width;
		w_offset += height*width;

		for (int z = 1; z < depth; z++)
		{
			if(!occupy[offset] && !occupy[offset-height*width])
				w[w_offset] = adv_w[w_offset];
			offset += height*width;
			w_offset += height*width;
		}
	}

	/****************************************************************************************/

	void cu_Copy_to_tex_velocity_regular(const float4* vel, hipArray** velocity_array)
	{
		tex_velocity_regular.normalized = true;
		tex_velocity_regular.filterMode = hipFilterModeLinear;
		tex_velocity_regular.addressMode[0] = hipAddressModeClamp;
		tex_velocity_regular.addressMode[1] = hipAddressModeClamp;
		tex_velocity_regular.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf4 = hipCreateChannelDesc<float4>();
		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth);

		checkCudaErrors( hipMalloc3DArray(velocity_array, &channelDescf4, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)vel, texSize.width*sizeof(float4), texSize.width, texSize.height);
		copyParams.dstArray = *velocity_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_velocity_regular,*velocity_array,channelDescf4) );
	}

	void cu_Free_tex_velocity_regular(hipArray** velocity_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_velocity_regular) );
		checkCudaErrors( hipFreeArray(*velocity_array) );
		*velocity_array = 0;
	}

	void cu_Copy_to_tex_velocity_MAC_u(const float* u, hipArray** u_array)
	{
		tex_velocity_MAC_u.normalized = true;
		tex_velocity_MAC_u.filterMode = hipFilterModeLinear;
		tex_velocity_MAC_u.addressMode[0] = hipAddressModeClamp;
		tex_velocity_MAC_u.addressMode[1] = hipAddressModeClamp;
		tex_velocity_MAC_u.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width+1,h_height,h_depth);

		checkCudaErrors( hipMalloc3DArray(u_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)u, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *u_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_velocity_MAC_u,*u_array,channelDescf) );
	}

	void cu_Free_tex_velocity_MAC_u(hipArray** u_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_velocity_MAC_u) );
		checkCudaErrors( hipFreeArray(*u_array) );
		*u_array = 0;
	}

	void cu_Copy_to_tex_velocity_MAC_v(const float* v, hipArray** v_array)
	{
		tex_velocity_MAC_v.normalized = true;
		tex_velocity_MAC_v.filterMode = hipFilterModeLinear;
		tex_velocity_MAC_v.addressMode[0] = hipAddressModeClamp;
		tex_velocity_MAC_v.addressMode[1] = hipAddressModeClamp;
		tex_velocity_MAC_v.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width,h_height+1,h_depth);

		checkCudaErrors( hipMalloc3DArray(v_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)v, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *v_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_velocity_MAC_v,*v_array,channelDescf) );
	}

	void cu_Free_tex_velocity_MAC_v(hipArray** v_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_velocity_MAC_v) );
		checkCudaErrors( hipFreeArray(*v_array) );
		*v_array = 0;
	}

	void cu_Copy_to_tex_velocity_MAC_w(const float* w, hipArray** w_array)
	{
		tex_velocity_MAC_w.normalized = true;
		tex_velocity_MAC_w.filterMode = hipFilterModeLinear;
		tex_velocity_MAC_w.addressMode[0] = hipAddressModeClamp;
		tex_velocity_MAC_w.addressMode[1] = hipAddressModeClamp;
		tex_velocity_MAC_w.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth+1);

		checkCudaErrors( hipMalloc3DArray(w_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)w, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *w_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_velocity_MAC_w,*w_array,channelDescf) );
	}

	void cu_Free_tex_velocity_MAC_w(hipArray** w_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_velocity_MAC_w) );
		checkCudaErrors( hipFreeArray(*w_array) );
		*w_array = 0;
	}

	void cu_Copy_to_tex_occupy(const bool* occupy, hipArray** occupy_array)
	{
		tex_occupy.normalized = true;                      
		tex_occupy.filterMode = hipFilterModePoint;     
		tex_occupy.addressMode[0] = hipAddressModeClamp; 
		tex_occupy.addressMode[1] = hipAddressModeClamp;
		tex_occupy.addressMode[2] = hipAddressModeClamp;	

		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth);
		hipChannelFormatDesc channelDescb = hipCreateChannelDesc<uchar1>();

		checkCudaErrors( hipMalloc3DArray(occupy_array, &channelDescb, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)occupy, texSize.width*sizeof(uchar1), texSize.width, texSize.height);
		copyParams.dstArray = *occupy_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_occupy,*occupy_array,channelDescb) );
	}

	void cu_Free_tex_occupy(hipArray** occupy_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_occupy) );
		checkCudaErrors( hipFreeArray(*occupy_array) );
		*occupy_array = 0;
	}

	void cu_Copy_to_tex_inputVelocity_regular(const float4* vel, hipArray** inputVelocity_array)
	{
		tex_inputVelocity_regular.normalized = true;
		tex_inputVelocity_regular.filterMode = hipFilterModeLinear;
		tex_inputVelocity_regular.addressMode[0] = hipAddressModeClamp;
		tex_inputVelocity_regular.addressMode[1] = hipAddressModeClamp;
		tex_inputVelocity_regular.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf4 = hipCreateChannelDesc<float4>();
		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth);

		checkCudaErrors( hipMalloc3DArray(inputVelocity_array, &channelDescf4, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)vel, texSize.width*sizeof(float4), texSize.width, texSize.height);
		copyParams.dstArray = *inputVelocity_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_inputVelocity_regular,*inputVelocity_array,channelDescf4) );
	}

	void cu_Free_tex_inputVelocity_regular(hipArray** inputVelocity_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_inputVelocity_regular) );
		checkCudaErrors( hipFreeArray(*inputVelocity_array) );
		*inputVelocity_array = 0;
	}

	void cu_Copy_to_tex_inputVelocity_MAC_u(const float* u, hipArray** u_array)
	{
		tex_inputVelocity_MAC_u.normalized = true;
		tex_inputVelocity_MAC_u.filterMode = hipFilterModeLinear;
		tex_inputVelocity_MAC_u.addressMode[0] = hipAddressModeClamp;
		tex_inputVelocity_MAC_u.addressMode[1] = hipAddressModeClamp;
		tex_inputVelocity_MAC_u.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width+1,h_height,h_depth);

		checkCudaErrors( hipMalloc3DArray(u_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)u, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *u_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_inputVelocity_MAC_u,*u_array,channelDescf) );
	}

	void cu_Free_tex_inputVelocity_MAC_u(hipArray** u_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_inputVelocity_MAC_u) );
		checkCudaErrors( hipFreeArray(*u_array) );
		*u_array = 0;
	}

	void cu_Copy_to_tex_inputVelocity_MAC_v(const float* v, hipArray** v_array)
	{
		tex_inputVelocity_MAC_v.normalized = true;
		tex_inputVelocity_MAC_v.filterMode = hipFilterModeLinear;
		tex_inputVelocity_MAC_v.addressMode[0] = hipAddressModeClamp;
		tex_inputVelocity_MAC_v.addressMode[1] = hipAddressModeClamp;
		tex_inputVelocity_MAC_v.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width,h_height+1,h_depth);

		checkCudaErrors( hipMalloc3DArray(v_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)v, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *v_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_inputVelocity_MAC_v,*v_array,channelDescf) );
	}

	void cu_Free_tex_inputVelocity_MAC_v(hipArray** v_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_inputVelocity_MAC_v) );
		checkCudaErrors( hipFreeArray(*v_array) );
		*v_array = 0;
	}

	void cu_Copy_to_tex_inputVelocity_MAC_w(const float* w, hipArray** w_array)
	{
		tex_inputVelocity_MAC_w.normalized = true;
		tex_inputVelocity_MAC_w.filterMode = hipFilterModeLinear;
		tex_inputVelocity_MAC_w.addressMode[0] = hipAddressModeClamp;
		tex_inputVelocity_MAC_w.addressMode[1] = hipAddressModeClamp;
		tex_inputVelocity_MAC_w.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth+1);

		checkCudaErrors( hipMalloc3DArray(w_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)w, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *w_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_inputVelocity_MAC_w,*w_array,channelDescf) );
	}

	void cu_Free_tex_inputVelocity_MAC_w(hipArray** w_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_inputVelocity_MAC_w) );
		checkCudaErrors( hipFreeArray(*w_array) );
		*w_array = 0;
	}

	void cu_Copy_to_tex_temperature(const float* temperature, hipArray** temperature_array)
	{
		tex_temperature.normalized = true;
		tex_temperature.filterMode = hipFilterModeLinear;
		tex_temperature.addressMode[0] = hipAddressModeClamp;
		tex_temperature.addressMode[1] = hipAddressModeClamp;
		tex_temperature.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth);

		checkCudaErrors( hipMalloc3DArray(temperature_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)temperature, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *temperature_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_temperature,*temperature_array,channelDescf) );
	}

	void cu_Free_tex_temperature(hipArray** temperature_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_temperature) );
		checkCudaErrors( hipFreeArray(*temperature_array) );
		*temperature_array = 0;
	}

	void cu_Copy_to_tex_density(const float* density, hipArray** density_array)
	{
		tex_density.normalized = true;
		tex_density.filterMode = hipFilterModeLinear;
		tex_density.addressMode[0] = hipAddressModeClamp;
		tex_density.addressMode[1] = hipAddressModeClamp;
		tex_density.addressMode[2] = hipAddressModeClamp;

		hipChannelFormatDesc channelDescf = hipCreateChannelDesc<float>();
		hipExtent texSize = make_hipExtent(h_width,h_height,h_depth);

		checkCudaErrors( hipMalloc3DArray(density_array, &channelDescf, texSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)density, texSize.width*sizeof(float), texSize.width, texSize.height);
		copyParams.dstArray = *density_array;
		copyParams.extent   = texSize;
		copyParams.kind     = hipMemcpyDeviceToDevice;
		checkCudaErrors( hipMemcpy3D(&copyParams) );

		checkCudaErrors( hipBindTextureToArray(tex_density,*density_array,channelDescf) );
	}

	void cu_Free_tex_density(hipArray** density_array)
	{
		checkCudaErrors( hipUnbindTexture(tex_density) );
		checkCudaErrors( hipFreeArray(*density_array) );
		*density_array = 0;
	}

	/**********************************************************************************/

	void cu_Velocity_Negative(float* u, float* v, float* w, int width, int height, int depth)
	{
		int u_len = (width+1)*height*depth;
		int v_len = width*(height+1)*depth;
		int w_len = width*height*(depth+1);
		dim3 blockSize(BLOCK_SIZE*BLOCK_SIZE,1);
		dim3 gridSize_u((u_len+blockSize.x-1)/blockSize.x,1);
		dim3 gridSize_v((v_len+blockSize.x-1)/blockSize.x,1);
		dim3 gridSize_w((w_len+blockSize.x-1)/blockSize.x,1);

		Velocity_Negative_Kernel<<<gridSize_u,blockSize>>>(u,u_len);
		Velocity_Negative_Kernel<<<gridSize_v,blockSize>>>(v,v_len);
		Velocity_Negative_Kernel<<<gridSize_w,blockSize>>>(w,w_len);
	}

	void cu_Velocity_Negative_4channels(float4* vel, int width, int height, int depth)
	{
		int vel_len = width*height*depth;
		dim3 blockSize(BLOCK_SIZE*BLOCK_SIZE,1);
		dim3 gridSize((vel_len+blockSize.x-1)/blockSize.x,1);

		Velocity_Negative_4channels_Kernel<<<gridSize,blockSize>>>(vel,vel_len);
	}

	void cu_Input_Increment(float* input, const float* input_star, int len)
	{
		dim3 blockSize(BLOCK_SIZE*BLOCK_SIZE,1);
		dim3 gridSize((len+blockSize.x-1)/blockSize.x,1);

		Input_Increment_Kernel<<<gridSize,blockSize>>>(input,input_star,len);
	}

	void cu_Input_Increment_4channels(float4* input, const float4* input_star, int len)
	{
		dim3 blockSize(BLOCK_SIZE*BLOCK_SIZE,1);
		dim3 gridSize((len+blockSize.x-1)/blockSize.x,1);

		Input_Increment_4channels_Kernel<<<gridSize,blockSize>>>(input,input_star,len);
	}

	void cu_Apply_Advect_Velocity_Result_Open(const float* adv_u, const float* adv_v, const float* adv_w, const bool* occupy, float* u, float* v, float* w)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize_u(((h_width+1)+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		dim3 gridSize_v((h_width+blockSize.x-1)/blockSize.x,((h_height+1)+blockSize.y-1)/blockSize.y);
		dim3 gridSize_w((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);

		 Apply_Advect_Velocity_Result_Open_u_Kernel<<<gridSize_u,blockSize>>>(adv_u,occupy,u,h_width,h_height,h_depth);
		 Apply_Advect_Velocity_Result_Open_v_Kernel<<<gridSize_v,blockSize>>>(adv_v,occupy,v,h_width,h_height,h_depth);
		 Apply_Advect_Velocity_Result_Open_w_Kernel<<<gridSize_w,blockSize>>>(adv_w,occupy,w,h_width,h_height,h_depth);
	}

	void cu_Apply_Advect_Velocity_Result_Closed(const float* adv_u, const float* adv_v, const float* adv_w, const bool* occupy, float* u, float* v, float* w)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize_u(((h_width+1)+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		dim3 gridSize_v((h_width+blockSize.x-1)/blockSize.x,((h_height+1)+blockSize.y-1)/blockSize.y);
		dim3 gridSize_w((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);

		 Apply_Advect_Velocity_Result_Closed_u_Kernel<<<gridSize_u,blockSize>>>(adv_u,occupy,u,h_width,h_height,h_depth);
		 Apply_Advect_Velocity_Result_Closed_v_Kernel<<<gridSize_v,blockSize>>>(adv_v,occupy,v,h_width,h_height,h_depth);
		 Apply_Advect_Velocity_Result_Closed_w_Kernel<<<gridSize_w,blockSize>>>(adv_w,occupy,w,h_width,h_height,h_depth);
	}

	void cu_Advect_Velocity_inRegular_outRegular(float* u, float* v, float* w, const bool* occupy, bool is_open)
	{
		float4* vel = 0;
		checkCudaErrors( hipMalloc((void**)&vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(u,v,w,vel,h_width,h_height,h_depth);
		
		hipArray* velocity_array = 0;
		hipArray* inputVelocity_array = 0;
		hipArray* occupy_array = 0;
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_inputVelocity_regular(vel,&inputVelocity_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Velocity_inRegular_outRegular_Kernel<<<gridSize,blockSize>>>(vel);

		float* out_u = 0;
		float* out_v = 0;
		float* out_w = 0;
		checkCudaErrors( hipMalloc((void**)&out_u,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMemset(out_u,0,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_v,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMemset(out_v,0,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_w,sizeof(float)*h_width*h_height*(h_depth+1)) );
		checkCudaErrors( hipMemset(out_w,0,sizeof(float)*h_width*h_height*(h_depth+1)) );
		
		ZQ_CUDA_MACtoRegular::cu_Regular4_to_MAC(vel,out_u,out_v,out_w,h_width,h_height,h_depth);

		if(is_open)
			cu_Apply_Advect_Velocity_Result_Open(out_u,out_v,out_w,occupy,u,v,w);
		else
			cu_Apply_Advect_Velocity_Result_Closed(out_u,out_v,out_w,occupy,u,v,w);

		checkCudaErrors( hipFree(vel) );
		checkCudaErrors( hipFree(out_u) );
		checkCudaErrors( hipFree(out_v) );
		checkCudaErrors( hipFree(out_w) );
		vel = 0;
		out_u = 0;
		out_v = 0;
		out_w = 0;

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);
		cu_Free_tex_occupy(&occupy_array);
	}

	void cu_Advect_Velocity_inRegular_outRegular_BFECC(float* u, float* v, float* w, const bool* occupy, bool is_open)
	{
		float4* vel = 0;
		checkCudaErrors( hipMalloc((void**)&vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(u,v,w,vel,h_width,h_height,h_depth);

		float4* in_out_vel = 0;
		checkCudaErrors( hipMalloc((void**)&in_out_vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(in_out_vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		
		hipArray* velocity_array = 0;
		hipArray* inputVelocity_array = 0;
		hipArray* occupy_array = 0;
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_inputVelocity_regular(vel,&inputVelocity_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Velocity_inRegular_outRegular_Kernel<<<gridSize,blockSize>>>(in_out_vel);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);

		cu_Velocity_Negative_4channels(vel,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_inputVelocity_regular(in_out_vel,&inputVelocity_array);
		Advect_Velocity_inRegular_outRegular_Kernel<<<gridSize,blockSize>>>(in_out_vel);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);

		cu_Velocity_Negative_4channels(vel,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Input_Increment_4channels(vel,in_out_vel,h_width*h_height*h_depth);
		cu_Copy_to_tex_inputVelocity_regular(vel,&inputVelocity_array);

		Advect_Velocity_inRegular_outRegular_Kernel<<<gridSize,blockSize>>>(in_out_vel);


		float* out_u = 0;
		float* out_v = 0;
		float* out_w = 0;
		checkCudaErrors( hipMalloc((void**)&out_u,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMemset(out_u,0,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_v,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMemset(out_v,0,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_w,sizeof(float)*h_width*h_height*(h_depth+1)) );
		checkCudaErrors( hipMemset(out_w,0,sizeof(float)*h_width*h_height*(h_depth+1)) );
		
		ZQ_CUDA_MACtoRegular::cu_Regular4_to_MAC(in_out_vel,out_u,out_v,out_w,h_width,h_height,h_depth);

		if(is_open)
			cu_Apply_Advect_Velocity_Result_Open(out_u,out_v,out_w,occupy,u,v,w);
		else
			cu_Apply_Advect_Velocity_Result_Closed(out_u,out_v,out_w,occupy,u,v,w);

		checkCudaErrors( hipFree(vel) );
		checkCudaErrors( hipFree(in_out_vel));
		checkCudaErrors( hipFree(out_u) );
		checkCudaErrors( hipFree(out_v) );
		checkCudaErrors( hipFree(out_w) );
		vel = 0;
		in_out_vel = 0;
		out_u = 0;
		out_v = 0;
		out_w = 0;

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);
		cu_Free_tex_occupy(&occupy_array);
	}

	void cu_Advect_Velocity_inMAC_outMAC(float* u, float* v, float* w, const bool* occupy, bool is_open)
	{
		hipArray* velocity_MAC_u_array = 0;
		hipArray* velocity_MAC_v_array = 0;
		hipArray* velocity_MAC_w_array = 0;
		hipArray* occupy_array = 0;
		hipArray* inputVelocity_MAC_u_array = 0;
		hipArray* inputVelocity_MAC_v_array = 0;
		hipArray* inputVelocity_MAC_w_array = 0;
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);
		cu_Copy_to_tex_inputVelocity_MAC_u(u,&inputVelocity_MAC_u_array);
		cu_Copy_to_tex_inputVelocity_MAC_v(v,&inputVelocity_MAC_v_array);
		cu_Copy_to_tex_inputVelocity_MAC_w(w,&inputVelocity_MAC_w_array);

		float* out_u = 0;
		float* out_v = 0;
		float* out_w = 0;
		checkCudaErrors( hipMalloc((void**)&out_u,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMemset(out_u,0,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_v,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMemset(out_v,0,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_w,sizeof(float)*h_width*h_height*(h_depth+1)) );
		checkCudaErrors( hipMemset(out_w,0,sizeof(float)*h_width*h_height*(h_depth+1)) );

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize_u(((h_width+1)+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		dim3 gridSize_v((h_width+blockSize.x-1)/blockSize.x,((h_height+1)+blockSize.y-1)/blockSize.y);
		dim3 gridSize_w((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Velocity_inMAC_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inMAC_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inMAC_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);
		
		if(is_open)
			cu_Apply_Advect_Velocity_Result_Open(out_u,out_v,out_w,occupy,u,v,w);
		else
			cu_Apply_Advect_Velocity_Result_Closed(out_u,out_v,out_w,occupy,u,v,w);

		checkCudaErrors( hipFree(out_u) );
		checkCudaErrors( hipFree(out_v) );
		checkCudaErrors( hipFree(out_w) );
		out_u = 0;
		out_v = 0;
		out_w = 0;

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_occupy(&occupy_array);
		cu_Free_tex_inputVelocity_MAC_u(&inputVelocity_MAC_u_array);
		cu_Free_tex_inputVelocity_MAC_v(&inputVelocity_MAC_v_array);
		cu_Free_tex_inputVelocity_MAC_w(&inputVelocity_MAC_w_array);
	}

	void cu_Advect_Velocity_inMAC_outMAC_BFECC(float* u, float* v, float* w, const bool* occupy, bool is_open)
	{
		hipArray* velocity_MAC_u_array = 0;
		hipArray* velocity_MAC_v_array = 0;
		hipArray* velocity_MAC_w_array = 0;
		hipArray* occupy_array = 0;
		hipArray* inputVelocity_MAC_u_array = 0;
		hipArray* inputVelocity_MAC_v_array = 0;
		hipArray* inputVelocity_MAC_w_array = 0;
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);
		cu_Copy_to_tex_inputVelocity_MAC_u(u,&inputVelocity_MAC_u_array);
		cu_Copy_to_tex_inputVelocity_MAC_v(v,&inputVelocity_MAC_v_array);
		cu_Copy_to_tex_inputVelocity_MAC_w(w,&inputVelocity_MAC_w_array);

		float* out_u = 0;
		float* out_v = 0;
		float* out_w = 0;
		checkCudaErrors( hipMalloc((void**)&out_u,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMemset(out_u,0,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_v,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMemset(out_v,0,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_w,sizeof(float)*h_width*h_height*(h_depth+1)) );
		checkCudaErrors( hipMemset(out_w,0,sizeof(float)*h_width*h_height*(h_depth+1)) );

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize_u(((h_width+1)+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		dim3 gridSize_v((h_width+blockSize.x-1)/blockSize.x,((h_height+1)+blockSize.y-1)/blockSize.y);
		dim3 gridSize_w((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Velocity_inMAC_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inMAC_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inMAC_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_inputVelocity_MAC_u(&inputVelocity_MAC_u_array);
		cu_Free_tex_inputVelocity_MAC_v(&inputVelocity_MAC_v_array);
		cu_Free_tex_inputVelocity_MAC_w(&inputVelocity_MAC_w_array);

		cu_Velocity_Negative(u,v,w,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Copy_to_tex_inputVelocity_MAC_u(out_u,&inputVelocity_MAC_u_array);
		cu_Copy_to_tex_inputVelocity_MAC_v(out_v,&inputVelocity_MAC_v_array);
		cu_Copy_to_tex_inputVelocity_MAC_w(out_w,&inputVelocity_MAC_w_array);

		Advect_Velocity_inMAC_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inMAC_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inMAC_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_inputVelocity_MAC_u(&inputVelocity_MAC_u_array);
		cu_Free_tex_inputVelocity_MAC_v(&inputVelocity_MAC_v_array);
		cu_Free_tex_inputVelocity_MAC_w(&inputVelocity_MAC_w_array);

		cu_Velocity_Negative(u,v,w,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Input_Increment(u,out_u,(h_width+1)*h_height*h_depth);
		cu_Input_Increment(v,out_v,h_width*(h_height+1)*h_depth);
		cu_Input_Increment(w,out_w,h_width*h_height*(h_depth+1));
		cu_Copy_to_tex_inputVelocity_MAC_u(u,&inputVelocity_MAC_u_array);
		cu_Copy_to_tex_inputVelocity_MAC_v(v,&inputVelocity_MAC_v_array);
		cu_Copy_to_tex_inputVelocity_MAC_w(w,&inputVelocity_MAC_w_array);

		Advect_Velocity_inMAC_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inMAC_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inMAC_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);
		
		if(is_open)
			cu_Apply_Advect_Velocity_Result_Open(out_u,out_v,out_w,occupy,u,v,w);
		else
			cu_Apply_Advect_Velocity_Result_Closed(out_u,out_v,out_w,occupy,u,v,w);

		checkCudaErrors( hipFree(out_u) );
		checkCudaErrors( hipFree(out_v) );
		checkCudaErrors( hipFree(out_w) );
		out_u = 0;
		out_v = 0;
		out_w = 0;

		cu_Free_tex_occupy(&occupy_array);
		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_inputVelocity_MAC_u(&inputVelocity_MAC_u_array);
		cu_Free_tex_inputVelocity_MAC_v(&inputVelocity_MAC_v_array);
		cu_Free_tex_inputVelocity_MAC_w(&inputVelocity_MAC_w_array);

	}

	void cu_Advect_Velocity_inRegular_outMAC(float* u, float* v, float* w, const bool* occupy, bool is_open)
	{
		float4* vel = 0;
		checkCudaErrors( hipMalloc((void**)&vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(u,v,w,vel,h_width,h_height,h_depth);

		hipArray* velocity_array = 0;
		hipArray* inputVelocity_array = 0;
		hipArray* occupy_array = 0;
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_inputVelocity_regular(vel,&inputVelocity_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);

		float* out_u = 0;
		float* out_v = 0;
		float* out_w = 0;
		checkCudaErrors( hipMalloc((void**)&out_u,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMemset(out_u,0,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_v,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMemset(out_v,0,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_w,sizeof(float)*h_width*h_height*(h_depth+1)) );
		checkCudaErrors( hipMemset(out_w,0,sizeof(float)*h_width*h_height*(h_depth+1)) );

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize_u(((h_width+1)+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		dim3 gridSize_v((h_width+blockSize.x-1)/blockSize.x,((h_height+1)+blockSize.y-1)/blockSize.y);
		dim3 gridSize_w((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Velocity_inRegular_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inRegular_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inRegular_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);
		
		if(is_open)
			cu_Apply_Advect_Velocity_Result_Open(out_u,out_v,out_w,occupy,u,v,w);
		else
			cu_Apply_Advect_Velocity_Result_Closed(out_u,out_v,out_w,occupy,u,v,w);

		checkCudaErrors( hipFree(vel) );
		checkCudaErrors( hipFree(out_u) );
		checkCudaErrors( hipFree(out_v) );
		checkCudaErrors( hipFree(out_w) );
		vel = 0;
		out_u = 0;
		out_v = 0;
		out_w = 0;

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);
		cu_Free_tex_occupy(&occupy_array);
	}

	void cu_Advect_Velocity_inRegular_outMAC_BFECC(float* u, float* v, float* w, const bool* occupy, bool is_open)
	{
		float4* vel = 0;
		float4* in_out_vel = 0;
		checkCudaErrors( hipMalloc((void**)&vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&in_out_vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(in_out_vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(u,v,w,vel,h_width,h_height,h_depth);

		hipArray* velocity_array = 0;
		hipArray* inputVelocity_array = 0;
		hipArray* occupy_array = 0;
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_inputVelocity_regular(vel,&inputVelocity_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);

		float* out_u = 0;
		float* out_v = 0;
		float* out_w = 0;
		checkCudaErrors( hipMalloc((void**)&out_u,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMemset(out_u,0,sizeof(float)*(h_width+1)*h_height*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_v,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMemset(out_v,0,sizeof(float)*h_width*(h_height+1)*h_depth) );
		checkCudaErrors( hipMalloc((void**)&out_w,sizeof(float)*h_width*h_height*(h_depth+1)) );
		checkCudaErrors( hipMemset(out_w,0,sizeof(float)*h_width*h_height*(h_depth+1)) );

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize_u(((h_width+1)+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		dim3 gridSize_v((h_width+blockSize.x-1)/blockSize.x,((h_height+1)+blockSize.y-1)/blockSize.y);
		dim3 gridSize_w((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Velocity_inRegular_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inRegular_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inRegular_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);
		
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(out_u,out_v,out_w,in_out_vel,h_width,h_height,h_depth);

		cu_Velocity_Negative_4channels(vel,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_inputVelocity_regular(in_out_vel,&inputVelocity_array);

		Advect_Velocity_inRegular_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inRegular_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inRegular_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);

		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(out_u,out_v,out_w,in_out_vel,h_width,h_height,h_depth);

		cu_Velocity_Negative_4channels(vel,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		
		cu_Input_Increment_4channels(vel,in_out_vel,h_width*h_height*h_depth);

		cu_Copy_to_tex_inputVelocity_regular(vel,&inputVelocity_array);

		Advect_Velocity_inRegular_outMAC_u_Kernel<<<gridSize_u,blockSize>>>(out_u);
		Advect_Velocity_inRegular_outMAC_v_Kernel<<<gridSize_v,blockSize>>>(out_v);
		Advect_Velocity_inRegular_outMAC_w_Kernel<<<gridSize_w,blockSize>>>(out_w);


		
		if(is_open)
			cu_Apply_Advect_Velocity_Result_Open(out_u,out_v,out_w,occupy,u,v,w);
		else
			cu_Apply_Advect_Velocity_Result_Closed(out_u,out_v,out_w,occupy,u,v,w);

		checkCudaErrors( hipFree(vel) );
		checkCudaErrors( hipFree(in_out_vel) );
		checkCudaErrors( hipFree(out_u) );
		checkCudaErrors( hipFree(out_v) );
		checkCudaErrors( hipFree(out_w) );
		vel = 0;
		in_out_vel = 0;
		out_u = 0;
		out_v = 0;
		out_w = 0;

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_inputVelocity_regular(&inputVelocity_array);
		cu_Free_tex_occupy(&occupy_array);
	}

	void cu_Advect_Scalar_Regular_Velocity(const float* u, const float* v, const float* w, const bool* occupy, const float* input_temperature, const float* input_density, 
								float* output_temperature, float* output_density)
	{
		
		float4* vel = 0;
		checkCudaErrors( hipMalloc((void**)&vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(u,v,w,vel,h_width,h_height,h_depth);

		hipArray* velocity_array = 0;
		hipArray* occupy_array = 0;
		hipArray* temperature_array = 0;
		hipArray* density_array = 0;
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);
		cu_Copy_to_tex_temperature(input_temperature,&temperature_array);
		cu_Copy_to_tex_density(input_density,&density_array);

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Scalar_Regular_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_occupy(&occupy_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);

		checkCudaErrors( hipFree(vel));
		vel = 0;
	}

	void cu_Advect_Scalar_Regular_Velocity_BFECC(const float* u, const float* v, const float* w, const bool* occupy, float* input_temperature, float* input_density, 
								float* output_temperature, float* output_density)
	{
		
		float4* vel = 0;
		checkCudaErrors( hipMalloc((void**)&vel,sizeof(float4)*h_width*h_height*h_depth) );
		checkCudaErrors( hipMemset(vel,0,sizeof(float4)*h_width*h_height*h_depth) );
		ZQ_CUDA_MACtoRegular::cu_MAC_to_Regular4(u,v,w,vel,h_width,h_height,h_depth);

		hipArray* velocity_array = 0;
		hipArray* occupy_array = 0;
		hipArray* temperature_array = 0;
		hipArray* density_array = 0;
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);
		cu_Copy_to_tex_temperature(input_temperature,&temperature_array);
		cu_Copy_to_tex_density(input_density,&density_array);

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Scalar_Regular_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);

		cu_Velocity_Negative_4channels(vel,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Copy_to_tex_temperature(output_temperature,&temperature_array);
		cu_Copy_to_tex_density(output_density,&density_array);

		Advect_Scalar_Regular_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);

		cu_Velocity_Negative_4channels(vel,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_regular(vel,&velocity_array);
		cu_Input_Increment(input_temperature,output_temperature,h_width*h_height*h_depth);
		cu_Input_Increment(input_density,output_density,h_width*h_height*h_depth);
		cu_Copy_to_tex_temperature(input_temperature,&temperature_array);
		cu_Copy_to_tex_density(input_density,&density_array);

		Advect_Scalar_Regular_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_regular(&velocity_array);
		cu_Free_tex_occupy(&occupy_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);

		checkCudaErrors( hipFree(vel));
		vel = 0;
	}

	void cu_Advect_Scalar_MAC_Velocity(const float* u, const float* v, const float* w, const bool* occupy, const float* input_temperature, const float* input_density, 
								float* output_temperature, float* output_density)
	{
		hipArray* velocity_MAC_u_array = 0;
		hipArray* velocity_MAC_v_array = 0;
		hipArray* velocity_MAC_w_array = 0;
		hipArray* occupy_array = 0;
		hipArray* temperature_array = 0;
		hipArray* density_array = 0;
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);
		cu_Copy_to_tex_temperature(input_temperature,&temperature_array);
		cu_Copy_to_tex_density(input_density,&density_array);

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Scalar_MAC_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_occupy(&occupy_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);
	}

	void cu_Advect_Scalar_MAC_Velocity_BFECC(float* u, float* v, float* w, const bool* occupy, float* input_temperature, float* input_density, 
								float* output_temperature, float* output_density)
	{
		hipArray* velocity_MAC_u_array = 0;
		hipArray* velocity_MAC_v_array = 0;
		hipArray* velocity_MAC_w_array = 0;
		hipArray* occupy_array = 0;
		hipArray* temperature_array = 0;
		hipArray* density_array = 0;
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Copy_to_tex_occupy(occupy,&occupy_array);
		cu_Copy_to_tex_temperature(input_temperature,&temperature_array);
		cu_Copy_to_tex_density(input_density,&density_array);

		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((h_width+blockSize.x-1)/blockSize.x,(h_height+blockSize.y-1)/blockSize.y);
		Advect_Scalar_MAC_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);

		cu_Velocity_Negative(u,v,w,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Copy_to_tex_temperature(output_temperature,&temperature_array);
		cu_Copy_to_tex_density(output_density,&density_array);

		Advect_Scalar_MAC_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);

		cu_Velocity_Negative(u,v,w,h_width,h_height,h_depth);
		cu_Copy_to_tex_velocity_MAC_u(u,&velocity_MAC_u_array);
		cu_Copy_to_tex_velocity_MAC_v(v,&velocity_MAC_v_array);
		cu_Copy_to_tex_velocity_MAC_w(w,&velocity_MAC_w_array);
		cu_Input_Increment(input_temperature,output_temperature,h_width*h_height*h_depth);
		cu_Input_Increment(input_density,output_density,h_width*h_height*h_depth);
		cu_Copy_to_tex_temperature(input_temperature,&temperature_array);
		cu_Copy_to_tex_density(input_density,&density_array);

		Advect_Scalar_MAC_Velocity_Kernel<<<gridSize,blockSize>>>(output_temperature,output_density);

		cu_Free_tex_velocity_MAC_u(&velocity_MAC_u_array);
		cu_Free_tex_velocity_MAC_v(&velocity_MAC_v_array);
		cu_Free_tex_velocity_MAC_w(&velocity_MAC_w_array);
		cu_Free_tex_occupy(&occupy_array);
		cu_Free_tex_temperature(&temperature_array);
		cu_Free_tex_density(&density_array);
	}

	/****************************************************************************************/

	extern "C"
	void ZQ_Cuda_Prepare_Advection(const unsigned int width, const unsigned int height, const unsigned int depth, const float voxelSize, const unsigned int steps, const float deltatt)
	{
		h_width = width;
		h_height = height;
		h_depth = depth;
		h_steps = steps;
		h_voxelSize = voxelSize;
		h_deltatt = deltatt;

		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ZQ_CUDA_Advection3D::d_width),&width,sizeof(unsigned int)));
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ZQ_CUDA_Advection3D::d_height),&height,sizeof(unsigned int)));
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ZQ_CUDA_Advection3D::d_depth),&depth,sizeof(unsigned int)));
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ZQ_CUDA_Advection3D::d_steps),&steps,sizeof(unsigned int)));
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ZQ_CUDA_Advection3D::d_voxelSize),&voxelSize,sizeof(float)));
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ZQ_CUDA_Advection3D::d_deltatt),&deltatt,sizeof(float)));	

		//int tmp = 0;
		//scanf("%d",&tmp);
		
		//checkCudaErrors( hipMemcpyFromSymbol(&h_width,HIP_SYMBOL(d_width),sizeof(unsigned int)));
		//checkCudaErrors( hipMemcpyFromSymbol(&h_height,HIP_SYMBOL(d_height),sizeof(unsigned int)));
		//checkCudaErrors( hipMemcpyFromSymbol(&h_steps,HIP_SYMBOL(d_steps),sizeof(unsigned int)));
		//checkCudaErrors( hipMemcpyFromSymbol(&h_voxelSize,HIP_SYMBOL(d_voxelSize),sizeof(float)));
		//checkCudaErrors( hipMemcpyFromSymbol(&h_deltatt,HIP_SYMBOL(d_deltatt),sizeof(float)));
		//printf("width = %d\n",h_width);
		//printf("height = %d\n",h_height);
		//printf("steps = %d\n",h_steps);
		//printf("voxelSize = %f\n",h_voxelSize);
		//printf("deltatt = %f\n",h_deltatt);
		
		//scanf("%d",&tmp);
	}

	extern "C"
	float Advect_Velocity(float* u, float* v, float* w, const bool* occupy, bool is_open, enum AdvectVelocityType type)
	{
		float time = 0;
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);

		/////
		float* d_u = 0;
		float* d_v = 0;
		float* d_w = 0;
		bool* d_occupy = 0;
		checkCudaErrors( hipMalloc((void**)&d_u,sizeof(float)*(h_width+1)*h_height*h_depth));
		checkCudaErrors( hipMalloc((void**)&d_v,sizeof(float)*h_width*(h_height+1)*h_depth));
		checkCudaErrors( hipMalloc((void**)&d_w,sizeof(float)*h_width*h_height*(h_depth+1)));
		checkCudaErrors( hipMalloc((void**)&d_occupy,sizeof(bool)*h_width*h_height*h_depth));
		checkCudaErrors( hipMemcpy(d_u,u,sizeof(float)*(h_width+1)*h_height*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMemcpy(d_v,v,sizeof(float)*h_width*(h_height+1)*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMemcpy(d_w,w,sizeof(float)*h_width*h_height*(h_depth+1),hipMemcpyHostToDevice));
		checkCudaErrors( hipMemcpy(d_occupy,occupy,sizeof(bool)*h_width*h_height*h_depth,hipMemcpyHostToDevice));

		switch(type)
		{
		case ADV_VEL_INREG_OUTREG:
			cu_Advect_Velocity_inRegular_outRegular(d_u,d_v,d_w,d_occupy,is_open);
			break;
		case ADV_VEL_INREG_OUTREG_BFECC:
			cu_Advect_Velocity_inRegular_outRegular_BFECC(d_u,d_v,d_w,d_occupy,is_open);
			break;
		case ADV_VEL_INMAC_OUTMAC:
			cu_Advect_Velocity_inMAC_outMAC(d_u,d_v,d_w,d_occupy,is_open);
			break;
		case ADV_VEL_INMAC_OUTMAC_BFECC:
			cu_Advect_Velocity_inMAC_outMAC_BFECC(d_u,d_v,d_w,d_occupy,is_open);
			break;
		case ADV_VEL_INREG_OUTMAC:
			cu_Advect_Velocity_inRegular_outMAC(d_u,d_v,d_w,d_occupy,is_open);
			break;
		case ADV_VEL_INREG_OUTMAC_BFECC:
			cu_Advect_Velocity_inRegular_outMAC_BFECC(d_u,d_v,d_w,d_occupy,is_open);
			break;
		}
		
		checkCudaErrors( hipMemcpy(u,d_u,sizeof(float)*(h_width+1)*h_height*h_depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(v,d_v,sizeof(float)*h_width*(h_height+1)*h_depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(w,d_w,sizeof(float)*h_width*h_height*(h_depth+1),hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(d_u) );
		checkCudaErrors( hipFree(d_v) );
		checkCudaErrors( hipFree(d_w) );
		checkCudaErrors( hipFree(d_occupy));
		d_u = 0;
		d_v = 0;
		d_w = 0;
		d_occupy = 0;
		
		hipEventRecord(stop,0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time,start,stop);
		return time;
	}

	extern "C"
	float Advect_Scalar(const float* u, const float* v, const float* w, const bool* occupy, const float* input_temperature, const float* input_density, 
					float* output_temperature, float* output_density, enum AdvectScalarType type)
	{
		float time = 0;
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);
		
		float* d_u = 0;
		float* d_v = 0;
		float* d_w = 0;
		bool* d_occupy = 0;
		float* d_input_temperature = 0;
		float* d_input_density = 0;
		float* d_output_temperature = 0;
		float* d_output_density = 0;
		checkCudaErrors( hipMalloc((void**)&d_u,sizeof(float)*(h_width+1)*h_height*h_depth));
		checkCudaErrors( hipMemcpy(d_u,u,sizeof(float)*(h_width+1)*h_height*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMalloc((void**)&d_v,sizeof(float)*h_width*(h_height+1)*h_depth));
		checkCudaErrors( hipMemcpy(d_v,v,sizeof(float)*h_width*(h_height+1)*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMalloc((void**)&d_w,sizeof(float)*h_width*h_height*(h_depth+1)));
		checkCudaErrors( hipMemcpy(d_w,w,sizeof(float)*h_width*h_height*(h_depth+1),hipMemcpyHostToDevice));
		checkCudaErrors( hipMalloc((void**)&d_occupy,sizeof(bool)*h_width*h_height*h_depth));
		checkCudaErrors( hipMemcpy(d_occupy,occupy,sizeof(bool)*h_width*h_height*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMalloc((void**)&d_input_temperature,sizeof(float)*h_width*h_height*h_depth));
		checkCudaErrors( hipMemcpy(d_input_temperature,input_temperature,sizeof(float)*h_width*h_height*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMalloc((void**)&d_input_density,sizeof(float)*h_width*h_height*h_depth));
		checkCudaErrors( hipMemcpy(d_input_density,input_density,sizeof(float)*h_width*h_height*h_depth,hipMemcpyHostToDevice));
		checkCudaErrors( hipMalloc((void**)&d_output_temperature,sizeof(float)*h_width*h_height*h_depth));
		checkCudaErrors( hipMemset(d_output_temperature,0,sizeof(float)*h_width*h_height*h_depth));
		checkCudaErrors( hipMalloc((void**)&d_output_density,sizeof(float)*h_width*h_height*h_depth));
		checkCudaErrors( hipMemset(d_output_density,0,sizeof(float)*h_width*h_height*h_depth));

		switch(type)
		{
		case ADV_SCA_MAC:
			cu_Advect_Scalar_MAC_Velocity(d_u,d_v,d_w,d_occupy,d_input_temperature,d_input_density,d_output_temperature,d_output_density);
			break;
		case ADV_SCA_REG:
			cu_Advect_Scalar_Regular_Velocity(d_u,d_v,d_w,d_occupy,d_input_temperature,d_input_density,d_output_temperature,d_output_density);
			break;
		}
		
		checkCudaErrors( hipMemcpy(output_temperature,d_output_temperature,sizeof(float)*h_width*h_height*h_depth,hipMemcpyDeviceToHost));
		checkCudaErrors( hipMemcpy(output_density,d_output_density,sizeof(float)*h_width*h_height*h_depth,hipMemcpyDeviceToHost));

		checkCudaErrors( hipFree(d_u));
		checkCudaErrors( hipFree(d_v));
		checkCudaErrors( hipFree(d_w));
		checkCudaErrors( hipFree(d_occupy));
		checkCudaErrors( hipFree(d_input_temperature));
		checkCudaErrors( hipFree(d_input_density));
		checkCudaErrors( hipFree(d_output_temperature));
		checkCudaErrors( hipFree(d_output_density));
		d_u = 0;
		d_v = 0;
		d_w = 0;
		d_occupy = 0;
		d_input_temperature = 0;
		d_input_density = 0;
		d_output_temperature = 0;
		d_output_density = 0;
		
		hipEventRecord(stop,0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time,start,stop);
		return time;
	}
}


#endif