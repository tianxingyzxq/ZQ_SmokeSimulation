#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_ADD_FORCE_3D_CU_
#define _ZQ_CUDA_ADD_FORCE_3D_CU_

#include "ZQlibCudaDefines.cuh"
#include "ZQ_CUDA_AddForce3D.cuh"
#include "ZQ_CUDA_AddForce3D.h"
#include "ZQ_CUDA_PoissonSolver3D.cuh"

namespace ZQ_CUDA_AddForce3D
{
	__global__
	void Compute_Vorticity_Vector_Scale_Kernel(float* vortVector, float* vortScale, const float* u, const float* v, const float* w, const float deltah, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width-1 || x <= 0 || y >= height-1 || y <= 0)
			return ;
		
		for(int z = 1; z < depth-1;z++)
		{
				int offset = z*height*width+y*width+x;
				vortVector[offset*3+0] = (w[offset+width] - w[offset-width] - v[offset+height*width] + v[offset-height*width])/(2*deltah);
				vortVector[offset*3+1] = (u[offset+height*width] - u[offset-height*width] - w[offset+1] + w[offset-1])/(2*deltah);
				vortVector[offset*3+2] = (v[offset+1] - v[offset-1] - u[offset+width] + u[offset-width])/(2*deltah);
				vortScale[offset] = sqrt(vortVector[offset*3+0]*vortVector[offset*3+0]
										+vortVector[offset*3+1]*vortVector[offset*3+1]
										+vortVector[offset*3+2]*vortVector[offset*3+2]);
		}	
	}
	
	
	__global__
	void Compute_Vorticity_Vector_Scale_Kernel(const int z, float* vortVector, float* vortScale, const float* u, const float* v, const float* w, const float deltah, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		x = x + 1;
		y = y + 1;
		if(x >= width-1|| y >= height-1) // x in [1,width-2],y in [1,height-2]
			return ;
		
		//z in [1,deoth-2] 
		
		int offset = z*height*width+y*width+x;
		vortVector[offset*3+0] = (w[offset+width] - w[offset-width] - v[offset+height*width] + v[offset-height*width])/(2*deltah);
		vortVector[offset*3+1] = (u[offset+height*width] - u[offset-height*width] - w[offset+1] + w[offset-1])/(2*deltah);
		vortVector[offset*3+2] = (v[offset+1] - v[offset-1] - u[offset+width] + u[offset-width])/(2*deltah);
		vortScale[offset] = sqrt(vortVector[offset*3+0]*vortVector[offset*3+0]
								+vortVector[offset*3+1]*vortVector[offset*3+1]
								+vortVector[offset*3+2]*vortVector[offset*3+2]);
		
	}
	
	__global__
	void Compute_Gradient_of_VorticityScale_Kernel(float* gradVort, float* vortScale, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
		
		for(int z = 0;z < depth;z++)
		{
			int offset = z*height*width+y*width+x;
			if(x == 0)
				gradVort[offset*3+0] = vortScale[offset+1] - vortScale[offset];
			else if(x == width-1)
				gradVort[offset*3+0] = vortScale[offset] - vortScale[offset-1];
			else
				gradVort[offset*3+0] = 0.5f*(vortScale[offset+1] - vortScale[offset-1]);

			if(y == 0)
				gradVort[offset*3+1] = vortScale[offset+width] - vortScale[offset];
			else if(y == height-1)
				gradVort[offset*3+1] = vortScale[offset] - vortScale[offset-width];
			else
				gradVort[offset*3+1] = 0.5f*(vortScale[offset+width] - vortScale[offset-width]);

			if(z == 0)
				gradVort[offset*3+2] = vortScale[offset+height*width] - vortScale[offset];
			else if(z == depth-1)
				gradVort[offset*3+2] = vortScale[offset] - vortScale[offset-height*width];
			else
				gradVort[offset*3+2] = 0.5f*(vortScale[offset+height*width] - vortScale[offset-height*width]);

			float len = sqrt(gradVort[offset*3+0]*gradVort[offset*3+0]
							+ gradVort[offset*3+1]*gradVort[offset*3+1]
							+ gradVort[offset*3+2]*gradVort[offset*3+2]);
			if(len != 0)
			{
				gradVort[offset*3+0] /= len;
				gradVort[offset*3+1] /= len;
				gradVort[offset*3+2] /= len;
			}
		}
	}
	
	__global__
	void Compute_Gradient_of_VorticityScale_Kernel(const int z, float* gradVort, float* vortScale, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
		
		// z in [0,depth-1]
		
		int offset = z*height*width+y*width+x;
		if(x == 0)
			gradVort[offset*3+0] = vortScale[offset+1] - vortScale[offset];
		else if(x == width-1)
			gradVort[offset*3+0] = vortScale[offset] - vortScale[offset-1];
		else
			gradVort[offset*3+0] = 0.5f*(vortScale[offset+1] - vortScale[offset-1]);

		if(y == 0)
			gradVort[offset*3+1] = vortScale[offset+width] - vortScale[offset];
		else if(y == height-1)
			gradVort[offset*3+1] = vortScale[offset] - vortScale[offset-width];
		else
			gradVort[offset*3+1] = 0.5f*(vortScale[offset+width] - vortScale[offset-width]);

		if(z == 0)
			gradVort[offset*3+2] = vortScale[offset+height*width] - vortScale[offset];
		else if(z == depth-1)
			gradVort[offset*3+2] = vortScale[offset] - vortScale[offset-height*width];
		else
			gradVort[offset*3+2] = 0.5f*(vortScale[offset+height*width] - vortScale[offset-height*width]);

		float len = sqrt(gradVort[offset*3+0]*gradVort[offset*3+0]
						+ gradVort[offset*3+1]*gradVort[offset*3+1]
						+ gradVort[offset*3+2]*gradVort[offset*3+2]);
		if(len != 0)
		{
			gradVort[offset*3+0] /= len;
			gradVort[offset*3+1] /= len;
			gradVort[offset*3+2] /= len;
		}
	}
	
	__global__
	void Compute_Force_Kernel(float* force, const float* gradVort, const float* vortVector, const float* temperature, const float confineCoeff, 
					const float buoyCoeff, const float deltah, const float Tamb, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
			
		for(int z = 0;z < depth;z++)
		{
			int offset = z*height*width+y*width+x;	
				
			force[offset*3+0] = confineCoeff*deltah*(gradVort[offset*3+1]*vortVector[offset*3+2]-gradVort[offset*3+2]*vortVector[offset*3+1]);
			force[offset*3+1] = confineCoeff*deltah*(gradVort[offset*3+2]*vortVector[offset*3+0]-gradVort[offset*3+0]*vortVector[offset*3+2]);
			force[offset*3+2] = confineCoeff*deltah*(gradVort[offset*3+0]*vortVector[offset*3+1]-gradVort[offset*3+1]*vortVector[offset*3+0]);
			force[offset*3+1] += buoyCoeff*(temperature[offset]-Tamb);
		}
	}	
	
	__global__
	void Compute_Force_Kernel(const int z, float* force, const float* gradVort, const float* vortVector, const float* temperature, const float confineCoeff, 
					const float buoyCoeff, const float deltah, const float Tamb, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
		
		//z in [0,depth-1]	
		
		int offset = z*height*width+y*width+x;	
				
		force[offset*3+0] = confineCoeff*deltah*(gradVort[offset*3+1]*vortVector[offset*3+2]-gradVort[offset*3+2]*vortVector[offset*3+1]);
		force[offset*3+1] = confineCoeff*deltah*(gradVort[offset*3+2]*vortVector[offset*3+0]-gradVort[offset*3+0]*vortVector[offset*3+2]);
		force[offset*3+2] = confineCoeff*deltah*(gradVort[offset*3+0]*vortVector[offset*3+1]-gradVort[offset*3+1]*vortVector[offset*3+0]);
		force[offset*3+1] += buoyCoeff*(temperature[offset]-Tamb);
		
	}
	
	__global__
	void AddForce_u_Kernel(float* mac_u, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x > width || y >= height)
			return ;
		
		for(int z = 0;z < depth;z++)
		{
			if(x == 0)
			{
				if(!occupy[z*height*width+y*width+x])
				{
					mac_u[z*height*(width+1)+y*(width+1)+x] += deltat*force[(z*height*width+y*width+x)*3];
				}
			}
			else if(x == width)
			{
				if(!occupy[z*height*width+y*width+x-1])
				{
					mac_u[z*height*(width+1)+y*(width+1)+x] += deltat*force[(z*height*width+y*width+x-1)*3];
				}
			}
			else
			{
				if(!occupy[z*height*width+y*width+x-1] && !occupy[z*height*width+y*width+x])
					mac_u[z*height*(width+1)+y*(width+1)+x] += deltat*0.5f*(force[(z*height*width+y*width+x)*3]+force[(z*height*width+y*width+x-1)*3]);
			}
		}
	}
	
	__global__
	void AddForce_u_Kernel(const int z, float* mac_u, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x > width || y >= height)
			return ;
		
		//z in [0,depth-1]
		
		if(x == 0)
		{
			if(!occupy[z*height*width+y*width+x])
			{
				mac_u[z*height*(width+1)+y*(width+1)+x] += deltat*force[(z*height*width+y*width+x)*3];
			}
		}
		else if(x == width)
		{
			if(!occupy[z*height*width+y*width+x-1])
			{
				mac_u[z*height*(width+1)+y*(width+1)+x] += deltat*force[(z*height*width+y*width+x-1)*3];
			}
		}
		else
		{
			if(!occupy[z*height*width+y*width+x-1] && !occupy[z*height*width+y*width+x])
				mac_u[z*height*(width+1)+y*(width+1)+x] += deltat*0.5f*(force[(z*height*width+y*width+x)*3]+force[(z*height*width+y*width+x-1)*3]);
		}
	}
	
	__global__
	void AddForce_v_Kernel(float* mac_v, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y > height)
			return ;
			
		for(int z = 0;z < depth;z++)
		{
			if(y == 0)
			{
				if(!occupy[z*height*width+y*width+x])
				{
					mac_v[z*(height+1)*width+y*width+x] += deltat*force[(z*height*width+y*width+x)*3+1];
				}
			}
			else if(y == height)
			{
				if(!occupy[z*height*width+(y-1)*width+x])
				{
					mac_v[z*(height+1)*width+y*width+x] += deltat*force[(z*height*width+(y-1)*width+x)*3+1];
				}
			}
			else
			{
				if(!occupy[z*height*width+(y-1)*width+x] && !occupy[z*height*width+y*width+x])
					mac_v[z*(height+1)*width+y*width+x] += deltat*0.5f*(force[(z*height*width+y*width+x)*3+1]+force[(z*height*width+(y-1)*width+x)*3+1]);
			}
		}
	}
	
	__global__
	void AddForce_v_Kernel(const int z, float* mac_v, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y > height)
			return ;
		
		//z in [0,depth-1]	
		if(y == 0)
		{
			if(!occupy[z*height*width+y*width+x])
			{
				mac_v[z*(height+1)*width+y*width+x] += deltat*force[(z*height*width+y*width+x)*3+1];
			}
		}
		else if(y == height)
		{
			if(!occupy[z*height*width+(y-1)*width+x])
			{
				mac_v[z*(height+1)*width+y*width+x] += deltat*force[(z*height*width+(y-1)*width+x)*3+1];
			}
		}
		else
		{
			if(!occupy[z*height*width+(y-1)*width+x] && !occupy[z*height*width+y*width+x])
				mac_v[z*(height+1)*width+y*width+x] += deltat*0.5f*(force[(z*height*width+y*width+x)*3+1]+force[(z*height*width+(y-1)*width+x)*3+1]);
		}
	}
	
	__global__
	void AddForce_w_Kernel(float* mac_w, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
			
		
		if(!occupy[y*width+x])
		{
			mac_w[y*width+x] += deltat*force[(y*width+x)*3+2];
		}
		
		if(!occupy[(depth-1)*height*width+y*width+x])
		{
			mac_w[depth*height*width+y*width+x] += deltat*force[((depth-1)*height*width+y*width+x)*3+2];
		}
			
		for(int z = 0;z < depth;z++)
		{
			if(!occupy[(z-1)*height*width+y*width+x] && !occupy[z*height*width+y*width+x])
				mac_w[z*height*width+y*width+x] += deltat*0.5f*(force[(z*height*width+y*width+x)*3+2]+force[((z-1)*height*width+y*width+x)*3+2]);
		}
	}
	
	__global__
	void AddForce_w_Kernel(const int z, float* mac_w, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
			
		if(z == 0)
		{
			if(!occupy[y*width+x])
			{
				mac_w[y*width+x] += deltat*force[(y*width+x)*3+2];

			}
		}
		else if(z == depth)
		{
			if(!occupy[(depth-1)*height*width+y*width+x])
			{
				mac_w[depth*height*width+y*width+x] += deltat*force[((depth-1)*height*width+y*width+x)*3+2];
			}
		}
		else
		{	
			if(!occupy[(z-1)*height*width+y*width+x] && !occupy[z*height*width+y*width+x])
				mac_w[z*height*width+y*width+x] += deltat*0.5f*(force[(z*height*width+y*width+x)*3+2]+force[((z-1)*height*width+y*width+x)*3+2]);
		}
	}
	
	
	
	/*****************************************************/

	void cu_Compute_Vorticity_Vector_Scale(float* vortVector, float* vortScale, float* u, float* v, float* w, const float deltah, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x, (height+blockSize.y-1)/blockSize.y);
		
		Compute_Vorticity_Vector_Scale_Kernel<<<gridSize,blockSize>>>(vortVector,vortScale,u,v,w,deltah,width,height,depth);
	}
	
	void cu_Compute_Vorticity_Vector_Scale2(float* vortVector, float* vortScale, float* u, float* v, float* w, const float deltah, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width-2+blockSize.x-1)/blockSize.x, (height-2+blockSize.y-1)/blockSize.y);
		
		for(int z = 1;z < depth-1;z++)
			Compute_Vorticity_Vector_Scale_Kernel<<<gridSize,blockSize>>>(z,vortVector,vortScale,u,v,w,deltah,width,height,depth);
	}
	
	void cu_Compute_Gradient_of_VorticityScale(float* gradVort, float* vortScale, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		Compute_Gradient_of_VorticityScale_Kernel<<<gridSize,blockSize>>>(gradVort, vortScale, width, height, depth);
	}
	
	void cu_Compute_Gradient_of_VorticityScale2(float* gradVort, float* vortScale, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		for(int z = 0;z < depth;z++)
			Compute_Gradient_of_VorticityScale_Kernel<<<gridSize,blockSize>>>(z, gradVort, vortScale, width, height, depth);
	}
	
	void cu_Compute_Force(float* force, const float* gradVort, const float* vortVector, const float* temperature, const float confineCoeff, const float buoyCoeff, 
			const float deltah, const float Tamb, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		Compute_Force_Kernel<<<gridSize,blockSize>>>(force,gradVort,vortVector,temperature,confineCoeff,buoyCoeff,deltah,Tamb,width,height,depth);
	}
	
	void cu_Compute_Force2(float* force, const float* gradVort, const float* vortVector, const float* temperature, const float confineCoeff, const float buoyCoeff, 
			const float deltah, const float Tamb, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		for(int z = 0;z < depth;z++)
			Compute_Force_Kernel<<<gridSize,blockSize>>>(z,force,gradVort,vortVector,temperature,confineCoeff,buoyCoeff,deltah,Tamb,width,height,depth);
	}
	
	void cu_AddForce_u_v_w(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		AddForce_u_Kernel<<<u_gridSize,blockSize>>>(mac_u,occupy,force,deltat,width,height,depth);
		AddForce_v_Kernel<<<v_gridSize,blockSize>>>(mac_v,occupy,force,deltat,width,height,depth);
		AddForce_w_Kernel<<<w_gridSize,blockSize>>>(mac_w,occupy,force,deltat,width,height,depth);
	}
	
	void cu_AddForce_u_v_w2(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const float* force, const float deltat, const int width, const int height, const int depth)
	{
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		for(int z = 0;z < depth;z++)
			AddForce_u_Kernel<<<u_gridSize,blockSize>>>(z,mac_u,occupy,force,deltat,width,height,depth);
		for(int z = 0;z < depth;z++)
			AddForce_v_Kernel<<<v_gridSize,blockSize>>>(z,mac_v,occupy,force,deltat,width,height,depth);
		
		for(int z = 0;z <= depth;z++)
			AddForce_w_Kernel<<<w_gridSize,blockSize>>>(z,mac_w,occupy,force,deltat,width,height,depth);
	}

	/****************************************************************/

	 
	void cu_AddForce3D(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const float* temperature, const float deltah, const float deltat, 
						const float buoyCoeff, const float confineCoeff, const float Tamb, const int width, const int height, const int depth)
	{	
		float* u = 0;
		float* v = 0;
		float* w = 0;
		checkCudaErrors( hipMalloc((void**)&u,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&v,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&w,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(u,0,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(v,0,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(w,0,sizeof(float)*width*height*depth) );
		
		ZQ_CUDA_PoissonSolver3D::cu_MAC_to_Regular_vel(u,v,w,mac_u,mac_v,mac_w,width,height,depth);
		
		float* vortVector = 0;
		float* vortScale = 0;
		checkCudaErrors( hipMalloc((void**)&vortVector,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMalloc((void**)&vortScale,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(vortVector,0,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMemset(vortScale,0,sizeof(float)*width*height*depth) );
		
		cu_Compute_Vorticity_Vector_Scale(vortVector,vortScale,u,v,w,deltah, width,height,depth);
		
		float* gradVort = 0;
		checkCudaErrors( hipMalloc((void**)&gradVort,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMemset(gradVort,0,sizeof(float)*width*height*depth*3) );
		
		cu_Compute_Gradient_of_VorticityScale(gradVort,vortScale,width,height,depth);	
		
		float* force = 0;
		checkCudaErrors( hipMalloc((void**)&force,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMemset(force,0,sizeof(float)*width*height*depth*3) );
		
		cu_Compute_Force(force,gradVort,vortVector, temperature, confineCoeff, buoyCoeff, deltah, Tamb, width, height, depth);
		
		cu_AddForce_u_v_w(mac_u,mac_v,mac_w,occupy,force,deltat,width,height,depth);
	
		checkCudaErrors( hipFree(u) );
		checkCudaErrors( hipFree(v) );
		checkCudaErrors( hipFree(w) );
		checkCudaErrors( hipFree(vortVector) );
		checkCudaErrors( hipFree(vortScale) );
		checkCudaErrors( hipFree(gradVort) );
		checkCudaErrors( hipFree(force) );
		
	}

	void cu_AddForce3D2(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const float* temperature, const float deltah, const float deltat, 
						const float buoyCoeff, const float confineCoeff, const float Tamb, const int width, const int height, const int depth)
	{	
		float* u = 0;
		float* v = 0;
		float* w = 0;
		checkCudaErrors( hipMalloc((void**)&u,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&v,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&w,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(u,0,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(v,0,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(w,0,sizeof(float)*width*height*depth) );
		
		ZQ_CUDA_PoissonSolver3D::cu_MAC_to_Regular_vel(u,v,w,mac_u,mac_v,mac_w,width,height,depth);
		
		float* vortVector = 0;
		float* vortScale = 0;
		checkCudaErrors( hipMalloc((void**)&vortVector,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMalloc((void**)&vortScale,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemset(vortVector,0,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMemset(vortScale,0,sizeof(float)*width*height*depth) );
		
		cu_Compute_Vorticity_Vector_Scale2(vortVector,vortScale,u,v,w,deltah, width,height,depth);
		
		float* gradVort = 0;
		checkCudaErrors( hipMalloc((void**)&gradVort,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMemset(gradVort,0,sizeof(float)*width*height*depth*3) );
		
		cu_Compute_Gradient_of_VorticityScale2(gradVort,vortScale,width,height,depth);	
		
		float* force = 0;
		checkCudaErrors( hipMalloc((void**)&force,sizeof(float)*width*height*depth*3) );
		checkCudaErrors( hipMemset(force,0,sizeof(float)*width*height*depth*3) );
		
		cu_Compute_Force2(force,gradVort,vortVector, temperature, confineCoeff, buoyCoeff, deltah, Tamb, width, height, depth);
		
		cu_AddForce_u_v_w2(mac_u,mac_v,mac_w,occupy,force,deltat,width,height,depth);
	
		checkCudaErrors( hipFree(u) );
		checkCudaErrors( hipFree(v) );
		checkCudaErrors( hipFree(w) );
		checkCudaErrors( hipFree(vortVector) );
		checkCudaErrors( hipFree(vortScale) );
		checkCudaErrors( hipFree(gradVort) );
		checkCudaErrors( hipFree(force) );
	}

	/**********************************************************/
	extern "C" 
	float AddForce3D(float* mac_u, float* mac_v, float* mac_w, const bool* occupy, const float* temperature, const float deltah, const float deltat, 
						const float buoyCoeff, const float confineCoeff, const float Tamb, const int width, const int height, const int depth, enum AddForceType type)
	{
		float time = 0;
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);
		
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		float* mac_w_d = 0;
		bool* occupy_d = 0;
		float* temperature_d = 0;
		
		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_w_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&occupy_d,sizeof(bool)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&temperature_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1)*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_w_d,mac_w,sizeof(float)*width*height*(depth+1),hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(occupy_d,occupy,sizeof(bool)*width*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(temperature_d,temperature,sizeof(float)*width*height*depth,hipMemcpyHostToDevice) );
		
		switch(type)
		{
		case AddForceType::ADD_FORCE_ENTIRE:
			cu_AddForce3D(mac_u_d, mac_v_d, mac_w_d, occupy_d, temperature_d, deltah, deltat, buoyCoeff, confineCoeff, Tamb, width, height, depth);
			break;
		case AddForceType::ADD_FORCE_SLICE:
			cu_AddForce3D(mac_u_d, mac_v_d, mac_w_d, occupy_d, temperature_d, deltah, deltat, buoyCoeff, confineCoeff, Tamb, width, height, depth);
			break;
		}
	
		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_w,mac_w_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToHost) );
		
		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(mac_w_d) );
		checkCudaErrors( hipFree(occupy_d) );
		checkCudaErrors( hipFree(temperature_d) );
		
		hipEventRecord(stop,0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time,start,stop);
		return time;
		
	}
}

#endif