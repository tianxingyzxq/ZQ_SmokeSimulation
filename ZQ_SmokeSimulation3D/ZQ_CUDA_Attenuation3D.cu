#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_ATTENUATION_3D_CU_
#define _ZQ_CUDA_ATTENUATION_3D_CU_

#include "ZQlibCudaDefines.cuh"

namespace ZQ_CUDA_Attenuation3D
{
	__global__
	void Atten_u_Kernel(float* mac_u, const bool* occupy, const float velAtten, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x > width || y >= height)
			return ;
		
		for(int z = 0;z < depth;z++)
		{
			if(x == 0)
			{
				if(!occupy[z*height*width+y*width+0])
					mac_u[z*height*(width+1)+y*(width+1)+0] *= velAtten;
			}
			else if(x == width)
			{
				if(!occupy[z*height*width+y*width+width-1])
					mac_u[z*height*(width+1)+y*(width+1)+width] *= velAtten;
			}
			else
			{
				if(!occupy[z*height*width+y*width+x-1] && !occupy[z*height*width+y*width+x])
					mac_u[z*height*(width+1)+y*(width+1)+x] *= velAtten;
			}
		}
	}
	
	__global__
	void Atten_v_Kernel(float* mac_v, const bool* occupy, const float velAtten, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y > height)
			return ;
		
		for(int z = 0;z < depth;z++)
		{
			if(y == 0)
			{
				if(!occupy[z*height*width+x])
					mac_v[z*(height+1)*width+x] *= velAtten;
			}
			else if(y == height)
			{
				if(!occupy[z*height*width+(height-1)*width+x])
					mac_v[z*(height+1)*width+height*width+x] *= velAtten;
			}
			else
			{
				if(!occupy[z*height*width+(y-1)*width+x] && !occupy[z*height*width+y*width+x])
					mac_v[z*(height+1)*width+y*width+x] *= velAtten;
			}
		}
	}
	
	__global__
	void Atten_w_Kernel(float* mac_w, const bool* occupy, const float velAtten, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
		
		if(!occupy[y*width+x])
			mac_w[y*width+x] *= velAtten;
		if(!occupy[(depth-1)*height*width+y*width+x])
			mac_w[depth*height*width+y*width+x] *= velAtten;
		for(int z = 1;z < depth;z++)
		{
			if(!occupy[(z-1)*height*width+y*width+x] && !occupy[z*height*width+y*width+x])
				mac_w[z*height*width+y*width+x] *= velAtten;
		}
	}
	
	__global__
	void Atten_temperature_density_Kernel(float* temperature, float* density, const float tempAtten, const float densityAtten, const int width, const int height, const int depth)
	{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int x = bx*blockDim.x+tx;
		int y = by*blockDim.y+ty;
		if(x >= width || y >= height)
			return ;
			
		for(int z = 0;z < depth;z++)
		{
			temperature[z*height*width+y*width+x] *= tempAtten;
			density[z*height*width+y*width+x] *= densityAtten;
		}
	}

	/*************************************************/

	void cu_Attenuation3D(float* mac_u, float* mac_v, float* mac_w, float* temperature, float* density, const bool* occupy, 
						const float velAtten, const float tempAtten, const float densityAtten, const int width, const int height, const int depth)
	{	
		dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
		dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 u_gridSize((width+1+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		dim3 v_gridSize((width+blockSize.x-1)/blockSize.x,(height+1+blockSize.y-1)/blockSize.y);
		dim3 w_gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y);
		
		Atten_u_Kernel<<<u_gridSize,blockSize>>>(mac_u,occupy,velAtten,width,height,depth);
		Atten_v_Kernel<<<v_gridSize,blockSize>>>(mac_v,occupy,velAtten,width,height,depth);
		Atten_w_Kernel<<<w_gridSize,blockSize>>>(mac_w,occupy,velAtten,width,height,depth);
		Atten_temperature_density_Kernel<<<gridSize,blockSize>>>(temperature,density,tempAtten,densityAtten,width,height,depth);
		
	}

	/***********************************************/

	extern "C"
	float Attenuation3D(float* mac_u, float* mac_v, float* mac_w, float* temperature, float* density, const bool* occupy, 
						const float velAtten, const float tempAtten, const float densityAtten, const int width, const int height, const int depth)
	{
		float time = 0;
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);
		
		float* mac_u_d = 0;
		float* mac_v_d = 0;
		float* mac_w_d = 0;
		float* temperature_d = 0;
		float* density_d = 0;
		bool* occupy_d = 0;
		
		checkCudaErrors( hipMalloc((void**)&mac_u_d,sizeof(float)*(width+1)*height*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_v_d,sizeof(float)*width*(height+1)*depth) );
		checkCudaErrors( hipMalloc((void**)&mac_w_d,sizeof(float)*width*height*(depth+1)) );
		checkCudaErrors( hipMalloc((void**)&temperature_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&density_d,sizeof(float)*width*height*depth) );
		checkCudaErrors( hipMalloc((void**)&occupy_d,sizeof(bool)*width*height*depth) );
		checkCudaErrors( hipMemcpy(mac_u_d,mac_u,sizeof(float)*(width+1)*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_v_d,mac_v,sizeof(float)*width*(height+1)*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(mac_w_d,mac_w,sizeof(float)*width*height*(depth+1),hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(temperature_d,temperature,sizeof(float)*width*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(density_d,density,sizeof(float)*width*height*depth,hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(occupy_d,occupy,sizeof(bool)*width*height*depth,hipMemcpyHostToDevice) );
		
		cu_Attenuation3D(mac_u_d,mac_v_d,mac_w_d,temperature_d,density_d,occupy_d,velAtten,tempAtten,densityAtten,width,height,depth);
		
		checkCudaErrors( hipMemcpy(mac_u,mac_u_d,sizeof(float)*(width+1)*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_v,mac_v_d,sizeof(float)*width*(height+1)*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(mac_w,mac_w_d,sizeof(float)*width*height*(depth+1),hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(density,density_d,sizeof(float)*width*height*depth,hipMemcpyDeviceToHost) );
		checkCudaErrors( hipMemcpy(temperature,temperature_d,sizeof(float)*width*height*depth,hipMemcpyDeviceToHost) );
		
		checkCudaErrors( hipFree(mac_u_d) );
		checkCudaErrors( hipFree(mac_v_d) );
		checkCudaErrors( hipFree(mac_w_d) );
		checkCudaErrors( hipFree(temperature_d) );
		checkCudaErrors( hipFree(density_d) );
		checkCudaErrors( hipFree(occupy_d) );
		mac_u_d = 0;
		mac_v_d = 0;
		mac_w_d = 0;
		temperature_d = 0;
		density_d = 0;
		occupy_d = 0;
		
		hipEventRecord(stop,0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time,start,stop);
		return time;
	}
}

#endif