#include "hip/hip_runtime.h"
#ifndef _ZQ_CUDA_POISSON_EDITING_3D_CU_
#define _ZQ_CUDA_POISSON_EDITING_3D_CU_


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

__global__
void poisson_editing3d_RedBlack_Kernel(const bool* mask, const float* laplace, float* output, const int width, const int height, const int depth, const int nChannels, bool redkernel)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int x = bx*blockDim.x+tx;
	int y = by*blockDim.y+ty;
	if(x >= width || y >= height)
		return ;

	int start_z = redkernel ? 0 : 1;

	int SLICE = width*height;
	for(int z = start_z;z < depth;z += 2)
	{
		int offset_single = z*height*width+y*width+x;
		for(int c = 0;c < nChannels;c++)
		{
			int offset = offset_single*nChannels+c;
			if(mask[offset])
			{
				float coeff = 6;
				float sigma = output[offset+SLICE] + output[offset-SLICE]
				+ output[offset+width] + output[offset-width]
				+ output[offset+1] + output[offset-1]
				- laplace[offset];
				output[offset] = sigma/coeff;
			}
		}
	}
}

////////////////////////
void cu_PoissonEditing3D(const bool* mask, const float* laplace, float* output, const int width, const int height, const int depth, const int nChannels, const int nIteration)
{
	dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE);
	dim3 gridSize((width+blockSize.x-1)/blockSize.x,(height+blockSize.y)/blockSize.y);

	for(int i = 0;i < nIteration;i++)
	{
		poisson_editing3d_RedBlack_Kernel<<<gridSize,blockSize>>>(mask,laplace,output,width,height,depth,nChannels,true);
		poisson_editing3d_RedBlack_Kernel<<<gridSize,blockSize>>>(mask,laplace,output,width,height,depth,nChannels,false);
	}
}

extern "C"
float ZQ_CUDA_PoissonEditing3D(const bool* mask, const float* laplace, float* output, const int width, const int height, const int depth, const int nChannels, const int nIteration)
{
	float time = 0;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	bool* mask_d = 0;
	float* laplace_d = 0;
	float* output_d = 0;
	checkCudaErrors( hipMalloc((void**)&mask_d,sizeof(bool)*width*height*depth) );
	checkCudaErrors( hipMalloc((void**)&laplace_d,sizeof(float)*width*height*depth*nChannels) );
	checkCudaErrors( hipMalloc((void**)&output_d,sizeof(float)*width*height*depth*nChannels) );

	checkCudaErrors( hipMemcpy(mask_d,mask,sizeof(bool)*width*height*depth,hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(laplace_d,laplace,sizeof(float)*width*height*depth*nChannels,hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(output_d,output,sizeof(float)*width*height*depth*nChannels,hipMemcpyHostToDevice) );

	cu_PoissonEditing3D(mask_d,laplace_d,output_d,width,height,depth,nChannels,nIteration);

	checkCudaErrors( hipMemcpy(output,output_d,sizeof(float)*width*height*depth*nChannels,hipMemcpyDeviceToHost) );

	checkCudaErrors( hipFree(mask_d) );
	checkCudaErrors( hipFree(laplace_d) );
	checkCudaErrors( hipFree(output_d) );
	mask_d = 0;
	laplace_d = 0;
	output_d = 0;

	hipEventRecord(stop,0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	return time;
}

#endif